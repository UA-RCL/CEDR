#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

__global__ void vector_add(const int* x, const int* y, int* z, int len) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < len) {
        z[id] = x[id] + y[id];
    }
}

extern "C" void Vector_Add_GPU(int** x, int** y, int** z, int* h_len) {
    printf("---------------------------------------\n");
    printf("------- Vector Addition on GPU --------\n");
    printf("---------------------------------------\n");
    const int length = *h_len;
    const int size_in_bytes = length * sizeof(int);
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;
    hipError_t err = hipSuccess;

    int* d_x = NULL;
    int* d_y = NULL;
    int* d_z = NULL;
    err = hipMalloc((void**)&d_x, size_in_bytes);
    err = hipMalloc((void**)&d_y, size_in_bytes);
    err = hipMalloc((void**)&d_z, size_in_bytes);

    err = hipMemcpy(d_x, (*x), size_in_bytes, hipMemcpyHostToDevice);
    err = hipMemcpy(d_y, (*y), size_in_bytes, hipMemcpyHostToDevice);

    
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_z, length);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vector_add kernel (error code %s)\n", hipGetErrorString(err));
    }

    hipMemcpy((*z), d_z, size_in_bytes, hipMemcpyDeviceToHost);

    hipFree(d_x); hipFree(d_y); hipFree(d_z);

    return;
}